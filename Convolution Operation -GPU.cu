/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__constant__ long int cfilter[2401];

__global__ void PerformCudaConvolutionOperation(long int*gmat,int m,int n,int k,long int*gans)
{
     __shared__ int s[5000];
     __shared__ int sr;
     __shared__ int er;
     __shared__ int sr2;
     __shared__ int nrb;
    
    long int id=blockIdx.x * blockDim.x+threadIdx.x;
    long int r=id/n;
    long int c=id%n;
    if(threadIdx.x==0)
    {
        sr=(r-k/2)<0?0:(r-k/2);
    }
    else if(threadIdx.x==1023)
    {
        er=(r+k/2)>m-1?m-1:(r+k/2);
    }
    else if(id==(m*n)-1)
      er=m-1;
    __syncthreads();
    nrb=er-sr+1;
    sr2=sr;
    if(threadIdx.x==0)
    {
        for(int i=0;sr<=er;i++,sr++)
        {
            for(int j=0;j<n;j++)
            s[i*n+j]=gmat[sr*n+j];
        }
    }
    __syncthreads();
    long int i,j,x,y;
     if(r==0)
     {
         x=k/2;
         i=r;
        if(c==0)
        {
             y=k/2;
             j=c;
        }
        else
        {
            y=(k/2-c)<0?0:(k/2-c);
            j=(c-k/2)<0?0:(c-k/2);
        }
    }
    else if(r>0)
    {
       x=(k/2-r)<0?0:(k/2-r);
       i=(r-k/2)<0?0:(r-k/2);
       if(c==0)
       {
          y=k/2;
          j=c;
       }
       
       else
       {
        j=(c-k/2)<0?0:(c-k/2);
        y=(k/2-c)<0?0:(k/2-c);
       }

    }
    i=i-sr2;
    long int sj=j;
    for(int p=x;p<k&&i<nrb;p++,i++)
    {
        for(int q=y;q<k&&j<n;q++,j++)
        gans[r*n+c]+= cfilter[p*k + q] * s[i * n + j];
        j=sj;

    }
    
}

int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    long int* gmat;
    long int* gans;
    hipMemcpyToSymbol(HIP_SYMBOL(cfilter), h_filter, k * k * sizeof(long int), 0, hipMemcpyHostToDevice);
    hipMalloc(&gmat,m*n*sizeof(long int));
    hipMemcpy(gmat,h_mat,m*n*sizeof(long int),hipMemcpyHostToDevice);
    hipMalloc(&gans,m*n*sizeof(long int));
    int blocks=ceil((m*n)/1024.0);

    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch
    PerformCudaConvolutionOperation<<<blocks,1024>>>(gmat,m,n,k,gans);
    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch
    hipMemcpy(h_ans,gans,m*n*sizeof(long int),hipMemcpyDeviceToHost);
    hipFree(gmat);
    hipFree(gans);
    
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */


    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}